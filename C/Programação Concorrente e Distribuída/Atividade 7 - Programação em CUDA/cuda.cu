/* Código realizado por:
- Thiago Hideki Sato (RA: 92341)
- Tiago Alexander Leal Ruzzon (RA: 92346)
- Tiago da Silva Cardoso (RA: 92347) */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define kappa 0.000045
#define T  0.01


__global__ void calculo (float *u, float *u_prev, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if (index < N + 1)
       u[index] = u_prev[index]+kappa*dt/(dx*dx)*(u_prev[index-1]-2*u_prev[index]+u_prev[index+1]);
}


int main(void) {
  float *tmp, *u, *u_prev; //Variáveis locais
  float *d_u, *d_u_prev; //PONTEIROS PARA CUDA
  float x, t, tempo;
  long int i, n, maxloc;
  hipEvent_t start, stop; //Cálculo do tempo
  int size_vet; //Tamanho do vetores
  int threads_por_bloco = 512;

  /* Calculando quantidade de pontos */
  n = tam/dx;
  size_vet = (n+1) * sizeof(float);

  hipEventCreate (&start);
  hipEventCreate (&stop);

  hipMalloc ((void**)&d_u, size_vet);
  hipMalloc ((void**)&d_u_prev, size_vet);

  /* Alocando vetores */
  u = (float *) malloc((n+1)*sizeof(float));
  u_prev = (float *) malloc((n+1)*sizeof(float));

  printf("Inicio: qtde=%ld, dt=%g, dx=%g, dx²=%g, kappa=%f, const=%f\n",
	 (n+1), dt, dx, dx*dx, kappa, kappa*dt/(dx*dx));
  printf("Iteracoes previstas: %g\n", T/dt);

  x = 0;
  for (i=0; i<n+1; i++) {
    if (x<=0.5) u_prev[i] = 200*x;
    else        u_prev[i] = 200*(1.0-x);
    x += dx;
  }

  printf("dx=%g, x=%g, x-dx=%g\n", dx, x, x-dx);
  printf("u_prev[0,1]=%g, %g\n",u_prev[0],u_prev[1]);
  printf("u_prev[n-1,n]=%g, %g\n",u_prev[n-1],u_prev[n]);

  t = 0.;

  hipEventRecord(start);
  while (t<T) {
    x = dx;

    hipMemcpy(d_u, u, size_vet, hipMemcpyHostToDevice);
    hipMemcpy(d_u_prev, u_prev, size_vet, hipMemcpyHostToDevice);

    calculo <<< (n + threads_por_bloco - 1) / threads_por_bloco, threads_por_bloco  >>> (d_u, d_u_prev, n);

    hipMemcpy(u, d_u, size_vet, hipMemcpyDeviceToHost);

    u[0] = u[n] = 0.; /* forca condicao de contorno */
    tmp = u_prev; u_prev = u; u = tmp; /* troca entre ponteiros */

    t += dt;
  }
  hipEventRecord(stop);

  //CALCULANDO O TEMPO
  hipEventSynchronize(stop);
  tempo = 0;
  hipEventElapsedTime(&tempo, start, stop);
  printf("\nNúmero de threads por bloco: %d\n", threads_por_bloco);
  printf("Tempo decorrido (parte paralelizada): %f ms\n\n", tempo);

  /* Calculando o maior valor e sua localizacao */
  maxloc = 0;
  for (i=1; i<n+1; i++) {
    if (u[i] > u[maxloc]) maxloc = i;
  }
  printf("Maior valor u[%ld] = %g\n", maxloc, u[maxloc]);

  return 0;
}
